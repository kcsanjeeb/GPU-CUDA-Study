#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Thread %d (block %d, thread %d) processing element %d\n",
           i, blockIdx.x, threadIdx.x, i);

    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 6;  // Small example
    size_t size = N * sizeof(float);

    // CPU arrays
    float h_A[] = {1, 2, 3, 4, 5, 6};
    float h_B[] = {10, 20, 30, 40, 50, 60};
    float h_C[6];

    // GPU arrays
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);    // Allocates GPU memory
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy to GPU // Transfer through PCIe bus
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);    // CPU→GPU
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);    // CPU→GPU

    // Launch kernel - 2 blocks, 4 threads/block (8 threads total)
    // Kernel runs on GPU, using GPU memory
    int threadsPerBlock = 4;
    int blocksPerGrid = 2;
    printf("Launching kernel with %d blocks, %d threads/block\n",
           blocksPerGrid, threadsPerBlock);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();  // Wait for GPU

    // Copy back
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print results
    printf("\nResults:\n");
    for (int i = 0; i < N; i++) {
        printf("C[%d] = %.1f + %.1f = %.1f\n", i, h_A[i], h_B[i], h_C[i]);
    }

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}